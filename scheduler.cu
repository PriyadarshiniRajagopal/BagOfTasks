#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include"task.h"
#include<sys/time.h>
#include <unistd.h>

struct timeval total_start;
struct timeval total_end;
int large_tasks=0;
int small_tasks=0;
int medium_tasks=0;
long total_time;
int *done_d;
int *done=0;
int global_taskId=0;
extern int M;
extern int T;
extern int sort;
int ttasks ;
int NBLOCKS = 6;
int done_sm[6]={0};
int *done_sm_d;
int *tt_array;
long *time_array;
int numtask[6]={0};
__device__ int F=0 ;
int current_tasks = 0;
//int count[6][100000];
int done_array[6][100000];
int persm[6]={0};
__device__ int current_index[6]={0};

long calcDiffTime(struct timeval* strtTime, struct timeval* endTime)
{
    return(
        endTime->tv_sec*1000000 + endTime->tv_usec
        - strtTime->tv_sec*1000000 - strtTime->tv_usec
        );
  
}


typedef void* (*op_func_t) (void*);
//Small task to be executed

__device__ void* fs(void *a)
{
int i;
    //printf("Started executing a small task\n");
    for(i=99;i>0;i--);
   // printf("Finished executing a small task\n");
    return NULL;
}

//Medium task to be executed
__device__ void* fm(void *a)
{
    //printf("Started executing a medium task\n");
    int b,c,d;
    for(b=99999;b>0;b--)
       {
        for(c=99999;c>0;c--)
           {
            for(d=99999;d>0;d--);
           }
       }
    for(b=99999;b>0;b--)
       {
        for(c=99999;c>0;c--)
           {
            for(d=99999;d>0;d--);
           }
       }
    for(b=99999;b>0;b--)
       {
        for(c=99999;c>0;c--)
           {
            for(d=99999;d>0;d--);
           }
       }
    for(b=99999;b>0;b--)
       {
        for(c=99999;c>0;c--)
           {
            for(d=99999;d>0;d--);
           }
       }
    for(b=99999;b>0;b--)
       {
        for(c=99999;c>0;c--)
           {
            for(d=99999;d>0;d--);
           }
       }
    //printf("Finished executing a medium task\n");
    return NULL;
}

// large task that's going to be executed

__device__ void* fl(void *a)
{
    //printf("Started executing a large task\n");
    long int k;
    long int i;
    for(k=99999999999999999;k>0;k--)
       {
        i++;
       }
    for(k=99999999999999999;k>0;k--)
       {
        i++;
       }
    long long int p;
    long long int q;
    for(p=999999999999999;p>0;p--)
       {
        for(q=9999999999999999;q>0;q--)
           i++;
       }
    for(p=9999999999999999;p>0;p--)
       {
        for(q=9999999999999999;q>0;q--)
           i++;
       }
    for(p=999999999999999;p>0;p--)
       {
        for(q=9999999999999999;q>0;q--)
           i++;
       }
    for(p=999999999999999;p>0;p--)
       {
        for(q=9999999999999999;q>0;q--)
           i++;
       }
    for(p=999999999999999;p>0;p--)
       {
        for(q=9999999999999999;q>0;q--)
           i++;
       }
    //printf("Finished executing a large task\n");
    return NULL;
}

//Static pointers to device functions

__device__ op_func_t p_fl = fl;
__device__ op_func_t p_fm = fm;
__device__ op_func_t p_fs = fs;

op_func_t h_fl;
op_func_t h_fm;
op_func_t h_fs;

//Utility function that prints the final time of the program
void printTasks()
{
total_time = calcDiffTime(&total_start,&total_end);
   printf("total time taken  = %ld\n",total_time);


}

//Function to generate a random number within the range
int rand_range(int min_n, int max_n)
{
    return rand() % (max_n - min_n + 1) + min_n;
}

//Function to add tasks to sms randomly
void randomAdd(int total,int smid)
{
   int s,x,sm;int type;
   //srand(1);
   
   int mf,lf;mf=lf=0;
   for (s=0;s<total;s++)
      {
       if(smid == -1)
          sm = s%6;
       else
          sm = smid;
          if(lf<total/3)
             {
              lf++;
              large_tasks++;
              type=2;
             }
           else if(mf<total/3)
             {
              mf++;
              medium_tasks++; 
              type=1;
             }
           else
             {
              type=0;
              small_tasks++;
             }
          //type=rand_range(0,2);
          //printf("The type of function=%d\n",type);
          if(type==0)
            x=taskAdd(h_fs,NULL,sm);
          else if(type==1)
            x=taskAdd(h_fm,NULL,sm);
          else
            x=taskAdd(h_fl,NULL,sm);
          
          tt_array[x] = type;
      }
}

//Fucntion to add tasks into specific SMs
void sortedAdd(int total,int smid)
{
    // printf("The number of tasks that are going to be added=%d\n",total);
     int s,x;
     int sm;
     for(s=0;s<total;s++)
        {
         if(smid == -1)
            sm = s%6;
         else
            sm = smid;
         if(sm == 0 || sm ==1)
            {
            //printf("Sm %d has small tasks\n",sm);
            x=taskAdd(h_fs,NULL,sm);
            tt_array[x] = 0;
            small_tasks++;
            }
         else if (sm == 2 || sm == 3)
           {
            //printf("Sm %d has medium tasks\n",sm);
            x=taskAdd(h_fm,NULL,sm);
            tt_array[x] = 1;
            medium_tasks++;
           }
         else if(sm == 4 ||sm ==5)
           {
            //printf("Sm %d has large tasks\n",sm);
            x=taskAdd(h_fl,NULL,sm);
            tt_array[x] = 2;
            large_tasks++;
           }
        } 

}
/*Utility function : to Compute the total running time of all tasks, min, max and average
void computeStats()
{
   int p;
   long total=0;
   long min = 999999999;
   long max = 0;
   double average;
   for(p=0;p<ttasks;p++)
      total+=time_array[p];   
   average = (total/ttasks);
   printf("The total running time of %d tasks = %ld\n",ttasks,total);
   printf("The average running time per task is = %f\n",average);

  for(p=0;p<ttasks;p++)
     {
      if(time_array[p] < min)
         min=time_array[p];
      if(time_array[p] > max)
         max=time_array[p];
     }

   printf("The minimum running time of a task is = %ld\n",min);
   printf("The maximin running time of a task is = %ld\n",max);

}
*/

//Utility function to get the type of the task-small 0,medium 1,large 2
int getTaskType(int taskId)
{
 return tt_array[taskId];

}

//Utility function to check is a task has completed execution or not
int taskDone(int taskId)
{
   int p,q;
   for(p=0;p<6;p++)
      {
       for(q=0;q<numtask[p];q++)
          {
           if(taskQueue[p].task_array[q].taskId == taskId && taskQueue[p].task_array[q].isComplete == 1)
              return TRUE;
          }
      }
   return FALSE;
}

//The API to add tasks to the bag
int taskAdd(void *(*func) (void *), void *arg, int sm)
{
    //printf("A new task is going to be added to a queue for smid %d\n",sm);
    int i=taskQueue[sm].i;
    //printf("Going to add to sm %d index %d\n",sm,i);
    int taskId;
    tasks[sm][i].taskId= global_taskId;
    taskId=global_taskId;
    //printf("the taskId of the task added is =%d\n",tasks[sm][i].taskId);
    global_taskId++;
    tasks[sm][i].func = func;
    tasks[sm][i].arg = arg;
    tasks[sm][i].isComplete=0;
    tasks[sm][i].isDisp = 0;
    taskQueue[sm].isDone=0;
    //gettimeofday(&tasks[sm][i].start,NULL);
    taskQueue[sm].task_array[i]=tasks[sm][i];
    (i)++;
    (taskQueue[sm].i)++;
    numtask[sm]++;
    persm[sm]++;
    return taskId;

}
//The functions required to get the smid of the thread
__device__ uint get_smid(void)
{
      uint ret;
      asm("mov.u32 %0, %smid;" : "=r"(ret) );
      return ret;
}

__global__ void kern(int *sm)
{
      if (threadIdx.x==0)
        sm[blockIdx.x]=get_smid();
}


/*
The scheduler:Once a thread enters the scheduler, the smid of that thread is computed so that the thread can pick up the tasks for that SM 
and execute the corresponding large, medium or small sized functions. After execution, the isComplete flag is set to 1 for that particular task so that the CPU can confirm the task as completed on polling in a busy wait loop.
*/
__global__ void scheduler(taskQueue_t *queues,int T,int *done_sm)
{
    int smid=get_smid();//gets the smid of the block-each thread from the same block will have the same smid value
    task_t task;//The task that's going to be executed next
       
    int index=-1 ;
    long int wa2=0;
    long int wa=99999;
    //int prev;
    index=atomicAdd(&current_index[smid],1);
    while(F<T)
       {
        wa++;
         while(wa>1)
           {
            wa--;
           }
         
         printf("");
         printf("");
         if(index<queues[smid].i)//keep executing
           {
             task=queues[smid].task_array[index];
                
                void * (*fp) (void *)=task.func;
                void *arg = task.arg;
                fp(arg);
                //Upon task completion, set isComplete flag for the task to 1
                queues[smid].task_array[index].isComplete=1;
                atomicAdd(&F,1);//Increment the value of F
                index=atomicAdd(&current_index[smid],1);
                if(index>=queues[smid].i)//
                   {
                    done_sm[smid]=1;
                    index=queues[smid].i;
                   }
           }
      if(wa2 < 999999)
         break;
      }
 
}
//Function to increment done value on the CPU side for threads of an sm
void setTime(int smid)
{
int c2=0;
while(persm[smid] >0)
{
 done_array[smid][c2]++;
 persm[smid]--;
 c2=(c2+1)%M;
}

}
/*
Function to allocate memory on the host and device for taskQueue and queues_d respectively. Also device function pointers that are only on the gpu side are copied to the host side using hipMemcpyFromSymbol.
*/
extern "C" void caller() {
    taskQueue_t *queues_d;//device copy 

    if(M>1000)
       M=1000;
    if(T>10000)
       T=10000;
    current_tasks = NBLOCKS * M;
    if(current_tasks > T)
       current_tasks=T;
    int r1,r2;
    for(r1=0;r1<6;r1++)
       {
        for(r2=0;r2<M;r2++)
           done_array[r1][r2]=0;
       }
    ttasks = T;//Initial number of tasks
    hipStream_t stream1,stream2,stream3;
    //done = 0;//needs to be changed on the gpu side
    tt_array = (int *) malloc(ttasks * sizeof(long));
    time_array = (long *)malloc(ttasks * sizeof(long));
    //done_sm = (int *)malloc(6 * sizeof(int));
    //start = (struct timeval *) malloc(ttasks * sizeof(struct timeval));
    //end = (struct timeval *) malloc(ttasks *sizeof(struct timeval));
    //Copy device function pointer to host side
    hipMemcpyFromSymbol(&h_fl, HIP_SYMBOL(p_fl), sizeof(op_func_t));
    hipMemcpyFromSymbol(&h_fm, HIP_SYMBOL(p_fm), sizeof(op_func_t));
    hipMemcpyFromSymbol(&h_fs, HIP_SYMBOL(p_fs), sizeof(op_func_t));
    //int s;
    int c1;
    int setComplete=0;
    //PIN the host memory! Very important for the Async memcpy function!
    hipHostMalloc( (void **) &taskQueue,sizeof(taskQueue_t)*6);
    hipHostMalloc( (void **) &done,sizeof(int));
    hipHostMalloc( (void **) &done_sm,sizeof(int)*6);
    //Allocate memory on CPU for taskQueue and tasks
    //taskQueue=(taskQueue_t*)calloc(6,sizeof(struct taskQueue));
    //tasks=(task_t **)calloc(6,sizeof(task_t*));
    int i;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    //for(i=0;i<6;i++)
       //tasks[i]=(task_t*)calloc(6*10000*NTHREADS*NBLOCKS,sizeof(task_t));
    for(i=0;i<6;i++)
       done_sm[i]=0;;    
    //Initialize all queues to have an initial index value of -1
    for(i=0;i<6;i++)
       taskQueue[i].i=0;
   
    //Adding tasks into the bag
     if(sort == 1)
        sortedAdd(current_tasks,-1);
     else
        randomAdd(current_tasks,-1);
    
    // Allocate memory on GPU
    hipMalloc( (void **) &queues_d,sizeof(taskQueue_t)*6);
    hipMalloc( (void **) &done_d, sizeof(int));
    hipMalloc( (void **) &done_sm_d,sizeof(int)*6);
    // copy from CPU to GPU
    hipMemcpy(done_sm_d,done_sm,sizeof(int)*6,hipMemcpyHostToDevice);
    hipMemcpy(queues_d,taskQueue,sizeof(taskQueue_t)*6,hipMemcpyHostToDevice);
    //hipMemcpy(done_d,done,sizeof(int),hipMemcpyHostToDevice);

     //for(s=0;s<ttasks;s++) 
     //      gettimeofday(&start[s],NULL);
     gettimeofday(&total_start,NULL);
//    for(s=0;s<M*NBLOCKS;s++)
//       gettimeofday(&start[s],NULL);
    //Kernel call to schedule all threads in the queue
    scheduler<<< NBLOCKS, M,0,stream1 >>>(queues_d,T,done_sm_d);
    int sm_num;
    long int wa2=0;
    // copy back from GPU to CPU
    while(setComplete<6)
       {
        wa2++;
        hipMemcpyAsync(done_sm,done_sm_d,sizeof(int)*6,hipMemcpyDeviceToHost,stream2);
        //hipMemcpyAsync(done,done_d,sizeof(int),hipMemcpyDeviceToHost,stream2);
        sm_num=-1;
        setComplete=0;
        for(c1=0;c1<6;c1++)
        {
        if(done_sm[c1] == 1)
           {
            setComplete++;
            setTime(c1);
            if(sm_num==-1)
               sm_num=c1;
           }
         }
        
        if(sm_num!=-1 && current_tasks < T)
           {
           for (r1=0;r1<6;r1++)
             persm[r1]=0;
             //printf("sm%d going to add more tasks\n",sm_num);
            if(sort == 1)
               {
                int rem=T-(M*NBLOCKS);
                rem =rem/2;
                sortedAdd(rem,sm_num);
                //printf("The sm to which more tasks were added %d\n",sm_num);
                current_tasks+=rem;
                sm_num=-1;
                setComplete=0;
               }
            else
               {
                int rem=T-(M*NBLOCKS);
                rem=rem/2;
                randomAdd(rem,sm_num);
                done_sm[sm_num]=0;
                //printf("The sm to which more tasks were added %d\n",sm_num);
                current_tasks+=rem;
                sm_num=-1;
                setComplete=0;
               }
           hipMemcpyAsync(queues_d,taskQueue,sizeof(taskQueue_t)*6,hipMemcpyHostToDevice,stream2);
           }
         if(wa2 >99999)
            break;
       }
   hipDeviceSynchronize();
    printf("Done array\n");
    for(r1=0;r1<6;r1++)
       {
        for(r2=0;r2<M;r2++)
           {
            printf("%d\t",done_array[r1][r2]);
           }
        printf("\n");
       }
    
     gettimeofday(&total_end,NULL);
       

      printTasks();
      if(setComplete!=6)
         printf("FAIL%\n");
      else
         printf("PASS \n");
    // free GPU memory
    hipFree(queues_d);
    hipFree(taskQueue);
    hipFree(done);
    hipFree(done_d);
    hipFree(done_sm);
    hipFree(done_sm_d);
}

